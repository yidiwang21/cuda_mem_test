#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <thread>
#include <sys/wait.h> 
#include <unistd.h>
#include <iostream>
#include <fstream>
#include <string>
#include "src/kernel.cu"
#include "src/support.cu"

using namespace std;

// for nvidia xavier
#define SM_NUM  8

int main (int argc, char *argv[]) {
    remove( "log_tranditional.txt" );
    std::string outfile = "log_tranditional.txt";

    hipError_t cuda_ret;


    float *A_h, *B_h, *C_h;
    float *A_d, *B_d, *C_d;
    size_t A_sz, B_sz, C_sz;
    unsigned VecSize = 10000000;

    int cp = fork();

    if (cp > 0) {
        A_sz = VecSize;
        B_sz = VecSize;
        C_sz = VecSize;
        A_h = (float*) malloc( sizeof(float)*A_sz );
        for (unsigned int i=0; i < A_sz; i++) { A_h[i] = (rand()%100)/100.00; }
    
        B_h = (float*) malloc( sizeof(float)*B_sz );
        for (unsigned int i=0; i < B_sz; i++) { B_h[i] = (rand()%100)/100.00; }
    
        C_h = (float*) malloc( sizeof(float)*C_sz );
        Timer timer;
    
        printf("Size Of vector: %u x %u\n  ", VecSize);
    
        // Allocate device variables ----------------------------------------------
    
        printf("Allocating device variables..."); fflush(stdout);
        startTime(&timer);
    
        //INSERT CODE HERE
        size_t bytes = sizeof(float) * VecSize;
        hipMalloc((void**) &A_d, bytes);
        hipMalloc((void**) &B_d, bytes);
        hipMalloc((void**) &C_d, bytes);
    
        hipDeviceSynchronize();
        stopTime(&timer); printf("%f s\n", elapsedTime(timer));
        std::string end_time0 = std::to_string(elapsedTime(timer) * 1000);
    
        // Copy host variables to device ------------------------------------------
    
        printf("Copying data from host to device..."); fflush(stdout);
    
        //INSERT CODE HERE
        hipMemcpy(A_d, A_h, bytes, hipMemcpyHostToDevice);
        hipMemcpy(B_d, B_h, bytes, hipMemcpyHostToDevice);
    
        hipDeviceSynchronize();
        stopTime(&timer); printf("%f s\n", elapsedTime(timer));
        std::string end_time1 = std::to_string(elapsedTime(timer) * 1000);
    
        // Launch kernel  ---------------------------
        basicVecAdd(A_d, B_d, C_d, VecSize); //In kernel.cu

        cuda_ret = hipDeviceSynchronize();
        if(cuda_ret != hipSuccess) {
            // FATAL("Unable to launch kernel");
            fprintf(stderr, "Unable to launch kernel\n");
            exit(-1);
        }
        stopTime(&timer); printf("%f s\n", elapsedTime(timer));
        std::string end_time2 = std::to_string(elapsedTime(timer) * 1000);

        // Copy device variables from host ----------------------------------------
    
        printf("Copying data from device to host..."); fflush(stdout);
    
        //INSERT CODE HERE
        hipMemcpy(C_h, C_d, bytes, hipMemcpyDeviceToHost);
    
        hipDeviceSynchronize();
        stopTime(&timer); printf("%f s\n", elapsedTime(timer));
        std::string end_time3 = std::to_string(elapsedTime(timer) * 1000);
    
    
    
        // Free memory ------------------------------------------------------------
    
        free(A_h);
        free(B_h);
        free(C_h);
    
        //INSERT CODE HERE
        hipFree(A_d);
        hipFree(B_d);
        hipFree(C_d);

        
        usleep(elapsedTime(timer) * 1000 * 1000 * 1);
        system("/home/nvidia/tegrastats --stop");
        stopTime(&timer); printf("%f s\n", elapsedTime(timer));
        std::string stop_time = std::to_string(elapsedTime(timer) * 1000);

        std::ofstream out;
        out.open(outfile, std::ios::app);
        out << "time0: " << end_time0 << std::endl;     // after alloc mem
        out << "time1: " << end_time1 << std::endl;     // after copied to device from host
        out << "time2: " << end_time2 << std::endl;     // after kernel execution
        out << "time3: " << end_time2 << std::endl;     // after copied to host from device
        out << "stoptime: " << stop_time << std::endl;  // after cooled down

        wait(NULL);
    }else if (cp == 0) {
        system(("/home/nvidia/tegrastats --interval 10 --logfile " + outfile).c_str());
    }


    return 0;

}