
#include <hip/hip_runtime.h>
#include <stdint.h>

__device__ inline uint64_t GlobalTimer64(void) {
    volatile uint64_t reading;
    asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(reading));
    return reading;
}

static __device__ __inline__ unsigned int GetSMID(void) {
    unsigned int ret;
    asm volatile("mov.u32 %0, %%smid;" : "=r"(ret));
    return ret;
}




static __global__ void kernel_func(uint64_t in) {
    uint64_t curr_time = 0;
    for (uint64_t i = 0; i < in; i++) {
        // may conflict when reading from the same reg concurrently
        // same results were gotten when I bypass this
        curr_time = GlobalTimer64();
     /* 
        int temp = temp % i;
        for (uint64_t j = 0; j < 2; j++) {
            temp %= j;
            temp += threadIdx.x;
        } */

    }
}